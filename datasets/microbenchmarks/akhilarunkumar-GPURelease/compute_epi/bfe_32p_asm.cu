#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
//#include <cutil.h>
#include <hip/hip_runtime.h>

float* h_A;
float* h_B;
float* h_C;
float* h_res;
float* d_A;
float* d_B;
float* d_C;
float* d_res;

__global__
//void compute(const float* A, const float* B, const float* C, float* D, int n) {
void compute(float* D, int n, int div) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    float I1 = tid * 2.0;

    int thread_id = threadIdx.x % 32;

    if (thread_id < div) {
        __asm volatile (
                " .reg .s32 %r111;\n\t"
                " .reg .s32 %r112;\n\t"
                " .reg .s32 %r113;\n\t"
                " .reg .s32 %r114;\n\t"
                " .reg .s32 %r115;\n\t"
                " .reg .s32 %r116;\n\t"
                " .reg .s32 %r117;\n\t"
                " .reg .s32 %r118;\n\t"
                " .reg .s32 %r119;\n\t"
                " .reg .s32 %r120;\n\t"
                " .reg .s32 %r121;\n\t"
                " .reg .s32 %r122;\n\t"
                " .reg .s32 %r123;\n\t"
                " .reg .s32 %r124;\n\t"
                " .reg .s32 %r125;\n\t"
                " .reg .s32 %r126;\n\t"
                " .reg .s32 %r127;\n\t"
                " .reg .s32 %r128;\n\t"
                "mov.s32 %r112, 44;\n\t"
                "mov.s32 %r113, %r112;\n\t"
                "mov.s32 %r114, 22;\n\t"
                "mov.s32 %r115, 33;\n\t"
                "mov.s32 %r116, 123;\n\t"
                "mov.s32 %r117, 242;\n\t"
                "mov.s32 %r118, 334;\n\t"
                "mov.s32 %r119, 562;\n\t"
                "mov.s32 %r120, 256;\n\t"
                "mov.s32 %r121, 156;\n\t"
                "mov.s32 %r122, 256;\n\t"
                "mov.s32 %r123, 556;\n\t"
                "mov.s32 %r124, 856;\n\t"
                "mov.s32 %r125, 356;\n\t"
                "mov.s32 %r126, 556;\n\t"
                "mov.s32 %r127, 656;\n\t"
                "mov.s32 %r128, 56;\n\t"

                );
        for (int k = 0; k < n; k++) {
            __asm volatile (
                    "bfe.s32 %r113, %r111, %r113, %r112;\n\t"
                    "bfe.s32 %r114, %r111, %r114, %r112;\n\t"
                    "bfe.s32 %r115, %r111, %r115, %r112;\n\t"
                    "bfe.s32 %r116, %r111, %r116, %r112;\n\t"
                    "bfe.s32 %r117, %r111, %r117, %r112;\n\t"
                    "bfe.s32 %r118, %r111, %r118, %r112;\n\t"
                    "bfe.s32 %r119, %r111, %r119, %r112;\n\t"
                    "bfe.s32 %r120, %r111, %r120, %r112;\n\t"
                    "bfe.s32 %r121, %r111, %r121, %r112;\n\t"
                    "bfe.s32 %r122, %r111, %r122, %r112;\n\t"
                    "bfe.s32 %r123, %r111, %r123, %r112;\n\t"
                    "bfe.s32 %r124, %r111, %r124, %r112;\n\t"
                    "bfe.s32 %r125, %r111, %r125, %r112;\n\t"
                    "bfe.s32 %r126, %r111, %r126, %r112;\n\t"
                    "bfe.s32 %r127, %r111, %r127, %r112;\n\t"
                    "bfe.s32 %r128, %r111, %r128, %r112;\n\t"
                    "bfe.s32 %r113, %r111, %r113, %r112;\n\t"
                    "bfe.s32 %r114, %r111, %r114, %r112;\n\t"
                    "bfe.s32 %r115, %r111, %r115, %r112;\n\t"
                    "bfe.s32 %r116, %r111, %r116, %r112;\n\t"
                    "bfe.s32 %r117, %r111, %r117, %r112;\n\t"
                    "bfe.s32 %r118, %r111, %r118, %r112;\n\t"
                    "bfe.s32 %r119, %r111, %r119, %r112;\n\t"
                    "bfe.s32 %r120, %r111, %r120, %r112;\n\t"
                    "bfe.s32 %r121, %r111, %r121, %r112;\n\t"
                    "bfe.s32 %r122, %r111, %r122, %r112;\n\t"
                    "bfe.s32 %r123, %r111, %r123, %r112;\n\t"
                    "bfe.s32 %r124, %r111, %r124, %r112;\n\t"
                    "bfe.s32 %r125, %r111, %r125, %r112;\n\t"
                    "bfe.s32 %r126, %r111, %r126, %r112;\n\t"
                    "bfe.s32 %r127, %r111, %r127, %r112;\n\t"
                    "bfe.s32 %r128, %r111, %r128, %r112;\n\t"
                    "bfe.s32 %r113, %r111, %r113, %r112;\n\t"
                    "bfe.s32 %r114, %r111, %r114, %r112;\n\t"
                    "bfe.s32 %r115, %r111, %r115, %r112;\n\t"
                    "bfe.s32 %r116, %r111, %r116, %r112;\n\t"
                    "bfe.s32 %r117, %r111, %r117, %r112;\n\t"
                    "bfe.s32 %r118, %r111, %r118, %r112;\n\t"
                    "bfe.s32 %r119, %r111, %r119, %r112;\n\t"
                    "bfe.s32 %r120, %r111, %r120, %r112;\n\t"
                    "bfe.s32 %r121, %r111, %r121, %r112;\n\t"
                    "bfe.s32 %r122, %r111, %r122, %r112;\n\t"
                    "bfe.s32 %r123, %r111, %r123, %r112;\n\t"
                    "bfe.s32 %r124, %r111, %r124, %r112;\n\t"
                    "bfe.s32 %r125, %r111, %r125, %r112;\n\t"
                    "bfe.s32 %r126, %r111, %r126, %r112;\n\t"
                    "bfe.s32 %r127, %r111, %r127, %r112;\n\t"
                    "bfe.s32 %r128, %r111, %r128, %r112;\n\t"
                    "bfe.s32 %r113, %r111, %r113, %r112;\n\t"
                    "bfe.s32 %r114, %r111, %r114, %r112;\n\t"
                    "bfe.s32 %r115, %r111, %r115, %r112;\n\t"
                    "bfe.s32 %r116, %r111, %r116, %r112;\n\t"
                    "bfe.s32 %r117, %r111, %r117, %r112;\n\t"
                    "bfe.s32 %r118, %r111, %r118, %r112;\n\t"
                    "bfe.s32 %r119, %r111, %r119, %r112;\n\t"
                    "bfe.s32 %r120, %r111, %r120, %r112;\n\t"
                    "bfe.s32 %r121, %r111, %r121, %r112;\n\t"
                    "bfe.s32 %r122, %r111, %r122, %r112;\n\t"
                    "bfe.s32 %r123, %r111, %r123, %r112;\n\t"
                    "bfe.s32 %r124, %r111, %r124, %r112;\n\t"
                    "bfe.s32 %r125, %r111, %r125, %r112;\n\t"
                    "bfe.s32 %r126, %r111, %r126, %r112;\n\t"
                    "bfe.s32 %r127, %r111, %r127, %r112;\n\t"
                    "bfe.s32 %r128, %r111, %r128, %r112;\n\t"
                    "bfe.s32 %r113, %r111, %r113, %r112;\n\t"
                    "bfe.s32 %r114, %r111, %r114, %r112;\n\t"
                    "bfe.s32 %r115, %r111, %r115, %r112;\n\t"
                    "bfe.s32 %r116, %r111, %r116, %r112;\n\t"
                    "bfe.s32 %r117, %r111, %r117, %r112;\n\t"
                    "bfe.s32 %r118, %r111, %r118, %r112;\n\t"
                    "bfe.s32 %r119, %r111, %r119, %r112;\n\t"
                    "bfe.s32 %r120, %r111, %r120, %r112;\n\t"
                    "bfe.s32 %r121, %r111, %r121, %r112;\n\t"
                    "bfe.s32 %r122, %r111, %r122, %r112;\n\t"
                    "bfe.s32 %r123, %r111, %r123, %r112;\n\t"
                    "bfe.s32 %r124, %r111, %r124, %r112;\n\t"
                    "bfe.s32 %r125, %r111, %r125, %r112;\n\t"
                    "bfe.s32 %r126, %r111, %r126, %r112;\n\t"
                    "bfe.s32 %r127, %r111, %r127, %r112;\n\t"
                    "bfe.s32 %r128, %r111, %r128, %r112;\n\t"
                    "bfe.s32 %r113, %r111, %r113, %r112;\n\t"
                    "bfe.s32 %r114, %r111, %r114, %r112;\n\t"
                    "bfe.s32 %r115, %r111, %r115, %r112;\n\t"
                    "bfe.s32 %r116, %r111, %r116, %r112;\n\t"
                    "bfe.s32 %r117, %r111, %r117, %r112;\n\t"
                    "bfe.s32 %r118, %r111, %r118, %r112;\n\t"
                    "bfe.s32 %r119, %r111, %r119, %r112;\n\t"
                    "bfe.s32 %r120, %r111, %r120, %r112;\n\t"
                    "bfe.s32 %r121, %r111, %r121, %r112;\n\t"
                    "bfe.s32 %r122, %r111, %r122, %r112;\n\t"
                    "bfe.s32 %r123, %r111, %r123, %r112;\n\t"
                    "bfe.s32 %r124, %r111, %r124, %r112;\n\t"
                    "bfe.s32 %r125, %r111, %r125, %r112;\n\t"
                    "bfe.s32 %r126, %r111, %r126, %r112;\n\t"
                    "bfe.s32 %r127, %r111, %r127, %r112;\n\t"
                    "bfe.s32 %r128, %r111, %r128, %r112;\n\t"
                    "bfe.s32 %r113, %r111, %r113, %r112;\n\t"
                    "bfe.s32 %r114, %r111, %r114, %r112;\n\t"
                    "bfe.s32 %r115, %r111, %r115, %r112;\n\t"
                    "bfe.s32 %r116, %r111, %r116, %r112;\n\t"
                    "bfe.s32 %r117, %r111, %r117, %r112;\n\t"
                    "bfe.s32 %r118, %r111, %r118, %r112;\n\t"
                    "bfe.s32 %r119, %r111, %r119, %r112;\n\t"
                    "bfe.s32 %r120, %r111, %r120, %r112;\n\t"
                    "bfe.s32 %r121, %r111, %r121, %r112;\n\t"
                    "bfe.s32 %r122, %r111, %r122, %r112;\n\t"
                    "bfe.s32 %r123, %r111, %r123, %r112;\n\t"
                    "bfe.s32 %r124, %r111, %r124, %r112;\n\t"
                    "bfe.s32 %r125, %r111, %r125, %r112;\n\t"
                    "bfe.s32 %r126, %r111, %r126, %r112;\n\t"
                    "bfe.s32 %r127, %r111, %r127, %r112;\n\t"
                    "bfe.s32 %r128, %r111, %r128, %r112;\n\t"
                    "bfe.s32 %r113, %r111, %r113, %r112;\n\t"
                    "bfe.s32 %r114, %r111, %r114, %r112;\n\t"
                    "bfe.s32 %r115, %r111, %r115, %r112;\n\t"
                    "bfe.s32 %r116, %r111, %r116, %r112;\n\t"
                    "bfe.s32 %r117, %r111, %r117, %r112;\n\t"
                    "bfe.s32 %r118, %r111, %r118, %r112;\n\t"
                    "bfe.s32 %r119, %r111, %r119, %r112;\n\t"
                    "bfe.s32 %r120, %r111, %r120, %r112;\n\t"
                    "bfe.s32 %r121, %r111, %r121, %r112;\n\t"
                    "bfe.s32 %r122, %r111, %r122, %r112;\n\t"
                    "bfe.s32 %r123, %r111, %r123, %r112;\n\t"
                    "bfe.s32 %r124, %r111, %r124, %r112;\n\t"
                    "bfe.s32 %r125, %r111, %r125, %r112;\n\t"
                    "bfe.s32 %r126, %r111, %r126, %r112;\n\t"
                    "bfe.s32 %r127, %r111, %r127, %r112;\n\t"
                    "bfe.s32 %r128, %r111, %r128, %r112;\n\t"
                    "bfe.s32 %r113, %r111, %r113, %r112;\n\t"
                    "bfe.s32 %r114, %r111, %r114, %r112;\n\t"
                    "bfe.s32 %r115, %r111, %r115, %r112;\n\t"
                    "bfe.s32 %r116, %r111, %r116, %r112;\n\t"
                    "bfe.s32 %r117, %r111, %r117, %r112;\n\t"
                    "bfe.s32 %r118, %r111, %r118, %r112;\n\t"
                    "bfe.s32 %r119, %r111, %r119, %r112;\n\t"
                    "bfe.s32 %r120, %r111, %r120, %r112;\n\t"
                    "bfe.s32 %r121, %r111, %r121, %r112;\n\t"
                    "bfe.s32 %r122, %r111, %r122, %r112;\n\t"
                    "bfe.s32 %r123, %r111, %r123, %r112;\n\t"
                    "bfe.s32 %r124, %r111, %r124, %r112;\n\t"
                    "bfe.s32 %r125, %r111, %r125, %r112;\n\t"
                    "bfe.s32 %r126, %r111, %r126, %r112;\n\t"
                    "bfe.s32 %r127, %r111, %r127, %r112;\n\t"
                    "bfe.s32 %r128, %r111, %r128, %r112;\n\t"
                    "bfe.s32 %r113, %r111, %r113, %r112;\n\t"
                    "bfe.s32 %r114, %r111, %r114, %r112;\n\t"
                    "bfe.s32 %r115, %r111, %r115, %r112;\n\t"
                    "bfe.s32 %r116, %r111, %r116, %r112;\n\t"
                    "bfe.s32 %r117, %r111, %r117, %r112;\n\t"
                    "bfe.s32 %r118, %r111, %r118, %r112;\n\t"
                    "bfe.s32 %r119, %r111, %r119, %r112;\n\t"
                    "bfe.s32 %r120, %r111, %r120, %r112;\n\t"
                    "bfe.s32 %r121, %r111, %r121, %r112;\n\t"
                    "bfe.s32 %r122, %r111, %r122, %r112;\n\t"
                    "bfe.s32 %r123, %r111, %r123, %r112;\n\t"
                    "bfe.s32 %r124, %r111, %r124, %r112;\n\t"
                    "bfe.s32 %r125, %r111, %r125, %r112;\n\t"
                    "bfe.s32 %r126, %r111, %r126, %r112;\n\t"
                    "bfe.s32 %r127, %r111, %r127, %r112;\n\t"
                    "bfe.s32 %r128, %r111, %r128, %r112;\n\t"
                    "bfe.s32 %r113, %r111, %r113, %r112;\n\t"
                    "bfe.s32 %r114, %r111, %r114, %r112;\n\t"
                    "bfe.s32 %r115, %r111, %r115, %r112;\n\t"
                    "bfe.s32 %r116, %r111, %r116, %r112;\n\t"
                    "bfe.s32 %r117, %r111, %r117, %r112;\n\t"
                    "bfe.s32 %r118, %r111, %r118, %r112;\n\t"
                    "bfe.s32 %r119, %r111, %r119, %r112;\n\t"
                    "bfe.s32 %r120, %r111, %r120, %r112;\n\t"
                    "bfe.s32 %r121, %r111, %r121, %r112;\n\t"
                    "bfe.s32 %r122, %r111, %r122, %r112;\n\t"
                    "bfe.s32 %r123, %r111, %r123, %r112;\n\t"
                    "bfe.s32 %r124, %r111, %r124, %r112;\n\t"
                    "bfe.s32 %r125, %r111, %r125, %r112;\n\t"
                    "bfe.s32 %r126, %r111, %r126, %r112;\n\t"
                    "bfe.s32 %r127, %r111, %r127, %r112;\n\t"
                    "bfe.s32 %r128, %r111, %r128, %r112;\n\t"
                    "bfe.s32 %r113, %r111, %r113, %r112;\n\t"
                    "bfe.s32 %r114, %r111, %r114, %r112;\n\t"
                    "bfe.s32 %r115, %r111, %r115, %r112;\n\t"
                    "bfe.s32 %r116, %r111, %r116, %r112;\n\t"
                    "bfe.s32 %r117, %r111, %r117, %r112;\n\t"
                    "bfe.s32 %r118, %r111, %r118, %r112;\n\t"
                    "bfe.s32 %r119, %r111, %r119, %r112;\n\t"
                    "bfe.s32 %r120, %r111, %r120, %r112;\n\t"
                    "bfe.s32 %r121, %r111, %r121, %r112;\n\t"
                    "bfe.s32 %r122, %r111, %r122, %r112;\n\t"
                    "bfe.s32 %r123, %r111, %r123, %r112;\n\t"
                    "bfe.s32 %r124, %r111, %r124, %r112;\n\t"
                    "bfe.s32 %r125, %r111, %r125, %r112;\n\t"
                    "bfe.s32 %r126, %r111, %r126, %r112;\n\t"
                    "bfe.s32 %r127, %r111, %r127, %r112;\n\t"
                    "bfe.s32 %r128, %r111, %r128, %r112;\n\t"
                    "bfe.s32 %r113, %r111, %r113, %r112;\n\t"
                    "bfe.s32 %r114, %r111, %r114, %r112;\n\t"
                    "bfe.s32 %r115, %r111, %r115, %r112;\n\t"
                    "bfe.s32 %r116, %r111, %r116, %r112;\n\t"
                    "bfe.s32 %r117, %r111, %r117, %r112;\n\t"
                    "bfe.s32 %r118, %r111, %r118, %r112;\n\t"
                    "bfe.s32 %r119, %r111, %r119, %r112;\n\t"
                    "bfe.s32 %r120, %r111, %r120, %r112;\n\t"
                    "bfe.s32 %r121, %r111, %r121, %r112;\n\t"
                    "bfe.s32 %r122, %r111, %r122, %r112;\n\t"
                    "bfe.s32 %r123, %r111, %r123, %r112;\n\t"
                    "bfe.s32 %r124, %r111, %r124, %r112;\n\t"
                    "bfe.s32 %r125, %r111, %r125, %r112;\n\t"
                    "bfe.s32 %r126, %r111, %r126, %r112;\n\t"
                    "bfe.s32 %r127, %r111, %r127, %r112;\n\t"
                    "bfe.s32 %r128, %r111, %r128, %r112;\n\t"
                    "bfe.s32 %r113, %r111, %r113, %r112;\n\t"
                    "bfe.s32 %r114, %r111, %r114, %r112;\n\t"
                    "bfe.s32 %r115, %r111, %r115, %r112;\n\t"
                    "bfe.s32 %r116, %r111, %r116, %r112;\n\t"
                    "bfe.s32 %r117, %r111, %r117, %r112;\n\t"
                    "bfe.s32 %r118, %r111, %r118, %r112;\n\t"
                    "bfe.s32 %r119, %r111, %r119, %r112;\n\t"
                    "bfe.s32 %r120, %r111, %r120, %r112;\n\t"
                    "bfe.s32 %r121, %r111, %r121, %r112;\n\t"
                    "bfe.s32 %r122, %r111, %r122, %r112;\n\t"
                    "bfe.s32 %r123, %r111, %r123, %r112;\n\t"
                    "bfe.s32 %r124, %r111, %r124, %r112;\n\t"
                    "bfe.s32 %r125, %r111, %r125, %r112;\n\t"
                    "bfe.s32 %r126, %r111, %r126, %r112;\n\t"
                    "bfe.s32 %r127, %r111, %r127, %r112;\n\t"
                    "bfe.s32 %r128, %r111, %r128, %r112;\n\t"
                    "bfe.s32 %r113, %r111, %r113, %r112;\n\t"
                    "bfe.s32 %r114, %r111, %r114, %r112;\n\t"
                    "bfe.s32 %r115, %r111, %r115, %r112;\n\t"
                    "bfe.s32 %r116, %r111, %r116, %r112;\n\t"
                    "bfe.s32 %r117, %r111, %r117, %r112;\n\t"
                    "bfe.s32 %r118, %r111, %r118, %r112;\n\t"
                    "bfe.s32 %r119, %r111, %r119, %r112;\n\t"
                    "bfe.s32 %r120, %r111, %r120, %r112;\n\t"
                    "bfe.s32 %r121, %r111, %r121, %r112;\n\t"
                    "bfe.s32 %r122, %r111, %r122, %r112;\n\t"
                    "bfe.s32 %r123, %r111, %r123, %r112;\n\t"
                    "bfe.s32 %r124, %r111, %r124, %r112;\n\t"
                    "bfe.s32 %r125, %r111, %r125, %r112;\n\t"
                    "bfe.s32 %r126, %r111, %r126, %r112;\n\t"
                    "bfe.s32 %r127, %r111, %r127, %r112;\n\t"
                    "bfe.s32 %r128, %r111, %r128, %r112;\n\t"
                    "bfe.s32 %r113, %r111, %r113, %r112;\n\t"
                    "bfe.s32 %r114, %r111, %r114, %r112;\n\t"
                    "bfe.s32 %r115, %r111, %r115, %r112;\n\t"
                    "bfe.s32 %r116, %r111, %r116, %r112;\n\t"
                    "bfe.s32 %r117, %r111, %r117, %r112;\n\t"
                    "bfe.s32 %r118, %r111, %r118, %r112;\n\t"
                    "bfe.s32 %r119, %r111, %r119, %r112;\n\t"
                    "bfe.s32 %r120, %r111, %r120, %r112;\n\t"
                    "bfe.s32 %r121, %r111, %r121, %r112;\n\t"
                    "bfe.s32 %r122, %r111, %r122, %r112;\n\t"
                    "bfe.s32 %r123, %r111, %r123, %r112;\n\t"
                    "bfe.s32 %r124, %r111, %r124, %r112;\n\t"
                    "bfe.s32 %r125, %r111, %r125, %r112;\n\t"
                    "bfe.s32 %r126, %r111, %r126, %r112;\n\t"
                    "bfe.s32 %r127, %r111, %r127, %r112;\n\t"
                    "bfe.s32 %r128, %r111, %r128, %r112;\n\t"
                    );
        }
    }
    __syncthreads();

    //    if ((blockDim.x * blockIdx.x + threadIdx.x) == 0)
    *D = I1;

    __syncthreads();
}

void usage() {
    std::cout << "Usage ./binary <num_blocks> <num_threads_per_block> <iterations>" "threads active per warp" << std::endl;
}

int main(int argc, char **argv)
{
    if (argc != 5) {
        usage();
        exit(1);
    }

    int num_blocks = atoi(argv[1]);
    int num_threads_per_block = atoi(argv[2]);
    int iterations = atoi(argv[3]);
    int divergence = atoi(argv[4]);

//    h_A = new float(2.0);
//    h_B = new float(3.0);
//    h_C = new float(4.0);

//    cudaMalloc((void**)&d_A, sizeof(float));
//    cudaMalloc((void**)&d_B, sizeof(float));
//    cudaMalloc((void**)&d_C, sizeof(float));
    hipMalloc((void**)&d_res, sizeof(float));

//    cudaMemcpy(d_A, h_A, sizeof(float), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_B, h_B, sizeof(float), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_C, h_C, sizeof(float), cudaMemcpyHostToDevice);

    hipEvent_t start, stop;
    float time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    hipProfilerStart();

//    compute<<<num_blocks, num_threads_per_block>>>(d_A, d_B, d_C, d_res, iterations);
    compute<<<num_blocks, num_threads_per_block>>>(d_res, iterations, divergence);

    hipProfilerStop();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);

    std::cout << "GPU Elapsed Time = " << time << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceSynchronize();

    hipMemcpy(h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);

    return 0;
}
