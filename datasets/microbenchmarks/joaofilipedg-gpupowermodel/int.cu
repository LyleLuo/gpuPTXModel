#include "hip/hip_runtime.h"
/*
 * Copyright 2011-2015 NVIDIA Corporation. All rights reserved
 *
 * Sample app to demonstrate use of CUPTI library to obtain metric values
 * using callbacks for CUDA runtime APIs
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
// #include <cupti.h>
#include <hip/hip_math_constants.h>
#include "lcutil.h"
#include <hip/hip_runtime_api.h>

#define ALIGN_SIZE (8)
#define ALIGN_BUFFER(buffer, align)                                            \
  (((uintptr_t) (buffer) & ((align)-1)) ? ((buffer) + (align) - ((uintptr_t) (buffer) & ((align)-1))) : (buffer))


#define COMP_ITERATIONS (512) //512
#define THREADS (1024)
#define BLOCKS (32760)
#define REGBLOCK_SIZE (4)
#define UNROLL_ITERATIONS (32)
#define deviceNum (0)

template <class T> __global__ void benchmark(){

	__shared__ T shared[THREADS];

	T r0 = shared[threadIdx.x],
	  r1 = r0,
	  r2 = r0,
	  r3 = r0;

	#pragma unroll 512
	for(int i=0; i<UNROLL_ITERATIONS; i++){
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
		r0 = r0 * r0 + r1;//r0;
		r1 = r1 * r1 + r2;//r1;
		r2 = r2 * r2 + r3;//r2;
		r3 = r3 * r3 + r0;//r3;
	}

	shared[threadIdx.x] = r0;

}

double median(int n, double x[][4],int col) {
    double temp;
    int i, j;
    // the following two loops sort the array x in ascending order
    for(i=0; i<n-1; i++) {
        for(j=i+1; j<n; j++) {
            if(x[j][col] < x[i][col]) {
                // swap elements
                temp = x[i][col];
                x[i][col] = x[j][col];
                x[j][col] = temp;
            }
        }
    }
    if(n%2==0) {
        // if there is an even number of elements, return mean of the two elements in the middle
        return((x[n/2][col] + x[n/2 - 1][col]) / 2.0);
    } else {
        // else return the element in the middle
        return x[n/2][col];
    }
}

void initializeEvents(hipEvent_t *start, hipEvent_t *stop){
	CUDA_SAFE_CALL( hipEventCreate(start) );
	CUDA_SAFE_CALL( hipEventCreate(stop) );
	CUDA_SAFE_CALL( hipEventRecord(*start, 0) );
}

float finalizeEvents(hipEvent_t start, hipEvent_t stop){
	CUDA_SAFE_CALL( hipGetLastError() );
	CUDA_SAFE_CALL( hipEventRecord(stop, 0) );
	CUDA_SAFE_CALL( hipEventSynchronize(stop) );
	float kernel_time;
	CUDA_SAFE_CALL( hipEventElapsedTime(&kernel_time, start, stop) );
	CUDA_SAFE_CALL( hipEventDestroy(start) );
	CUDA_SAFE_CALL( hipEventDestroy(stop) );
	return kernel_time;
}

void runbench(int type, double* kernel_time, double* flops){

	const long long computations = 2*(long long)(COMP_ITERATIONS)*REGBLOCK_SIZE*THREADS*BLOCKS;

	dim3 dimBlock(THREADS, 1, 1);
    dim3 dimGrid(BLOCKS, 1, 1);
	hipEvent_t start, stop;

	initializeEvents(&start, &stop);

    benchmark<int><<< dimGrid, dimBlock >>>();

	hipDeviceSynchronize();

	double time = finalizeEvents(start, stop);
	double result = ((double)computations)/(double)time*1000./(double)(1000*1000*1000);

	*kernel_time = time;
	*flops=result;

}

int main(int argc, char *argv[]){
	// CUpti_SubscriberHandle subscriber;
	// hipCtx_t context = 0;
	hipDevice_t device = 0;
	int deviceCount;
	// cupti_eventData cuptiEvent;
	// RuntimeApiTrace_t trace;


	printf("Usage: %s [device_num] [metric_name]\n", argv[0]);
	int ntries;
	if (argc>1){
		ntries = atoi(argv[1]);
	}else{
		ntries = 1;
	}

	hipSetDevice(deviceNum);
	double time[ntries][2],value[ntries][4];

	(hipGetDeviceCount(&deviceCount));
	if (deviceCount == 0) {
		printf("There is no device supporting CUDA.\n");
	return -2;
	}

	printf("CUDA Device Number: %d\n", deviceNum);

	(hipDeviceGet(&device, deviceNum));
	// CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, device));
	int measure;
	(hipDeviceGetAttribute(&measure,hipDeviceAttributeMaxThreadsPerBlock,device));
	printf("Max Threads per block = %d\n", measure);


	// DRIVER_API_CALL(hipCtxCreate(&context, 0, device));
	int i;

	for (i=0;i<ntries;i++){

		runbench(0,&time[0][0],&value[0][0]);


		printf("Registered time: %f ms\n",time[0][0]);

	}




	CUDA_SAFE_CALL( hipDeviceReset());

	return 0;
}
