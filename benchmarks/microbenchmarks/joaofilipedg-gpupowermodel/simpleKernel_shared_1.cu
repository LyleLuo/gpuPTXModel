// Utilities and system includes
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
// #include <cupti.h>
#include <hip/hip_runtime_api.h>

#define DATA_TYPE 0 // 0-SP, 1-INT, 2-DP
#define THREADS 1024

#define TILE_DIM 1024
#define SIZE 60000000

#define INNER_REPS 1

template <class T> __global__ void simpleKernel2()
{
    __shared__ T shared[THREADS];
    T r0;
    int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;


    if (xIndex < SIZE) {
        #pragma unroll 1
        for (int i=0;i<INNER_REPS;i++) {
            r0 = shared[threadIdx.x];
            shared[THREADS - threadIdx.x - 1] = r0;
        }
    }
}


int main(int argc, char **argv) {
    int inner_reps, outer_reps, vector_size, tile_dim;
    inner_reps = INNER_REPS;
    vector_size = SIZE;
    tile_dim = TILE_DIM;

    if (argc>1){
        outer_reps = atoi(argv[1]);
    }else{
        outer_reps = 1;
    }

    // execution configuration parameters
    dim3 grid(vector_size/tile_dim, 1), threads(tile_dim, 1);

    // CUDA events
    hipEvent_t start, stop;

    // print out common data for all kernels
    printf("\nVector size: %d  TotalBlocks: %d blockSize: %d\n\n", vector_size, grid.x, threads.x);

    // initialize events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // take measurements for loop over kernel launches
    hipEventRecord(start, 0);

    for (int i=0; i < outer_reps; i++)
    {
        simpleKernel2<float><<<grid, threads>>>();
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float kernelTime;
    hipEventElapsedTime(&kernelTime, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceReset();

    printf("Test passed\n");

    exit(EXIT_SUCCESS);
}
