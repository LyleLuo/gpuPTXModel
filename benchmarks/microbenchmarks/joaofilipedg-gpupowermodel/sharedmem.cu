#include "hip/hip_runtime.h"
/*
 * Copyright 2011-2015 NVIDIA Corporation. All rights reserved
 *
 * Sample app to demonstrate use of CUPTI library to obtain metric values
 * using callbacks for CUDA runtime APIs
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
// #include <cupti.h>
#include <hip/hip_math_constants.h>
#include "lcutil.h"
#include <hip/hip_runtime_api.h>
// #include <gpuCUPTISampler.cuh>

#define METRIC_NAME_TESLA "branch_efficiency"
#define METRIC_NAME_FERMI "ipc"


#define ALIGN_SIZE (8)
#define ALIGN_BUFFER(buffer, align)                                            \
  (((uintptr_t) (buffer) & ((align)-1)) ? ((buffer) + (align) - ((uintptr_t) (buffer) & ((align)-1))) : (buffer))


  #define COMP_ITERATIONS (16384) //k40c
  // #define COMP_ITERATIONS (32768) // titanx
// #define THREADS (1024)
// #define BLOCKS (1024)
#define THREADS (1024)
#define BLOCKS (32760)
#define REGBLOCK_SIZE (4)
#define UNROLL_ITERATIONS (32)
#define deviceNum (0)

template <class T> __global__ void benchmark (){

	__shared__ T shared[THREADS];


	T r0;

    #pragma unroll 16384
	for(int j=0; j<COMP_ITERATIONS; j+=UNROLL_ITERATIONS){
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
    	r0 = shared[threadIdx.x];
    	shared[THREADS - threadIdx.x - 1] = r0;
	}
}

double median(int n, double x[][4],int col) {
    double temp;
    int i, j;
    // the following two loops sort the array x in ascending order
    for(i=0; i<n-1; i++) {
        for(j=i+1; j<n; j++) {
            if(x[j][col] < x[i][col]) {
                // swap elements
                temp = x[i][col];
                x[i][col] = x[j][col];
                x[j][col] = temp;
            }
        }
    }
    if(n%2==0) {
        // if there is an even number of elements, return mean of the two elements in the middle
        return((x[n/2][col] + x[n/2 - 1][col]) / 2.0);
    } else {
        // else return the element in the middle
        return x[n/2][col];
    }
}

void initializeEvents(hipEvent_t *start, hipEvent_t *stop){
	CUDA_SAFE_CALL( hipEventCreate(start) );
	CUDA_SAFE_CALL( hipEventCreate(stop) );
	CUDA_SAFE_CALL( hipEventRecord(*start, 0) );
}

float finalizeEvents(hipEvent_t start, hipEvent_t stop){
	CUDA_SAFE_CALL( hipGetLastError() );
	CUDA_SAFE_CALL( hipEventRecord(stop, 0) );
	CUDA_SAFE_CALL( hipEventSynchronize(stop) );
	float kernel_time;
	CUDA_SAFE_CALL( hipEventElapsedTime(&kernel_time, start, stop) );
	CUDA_SAFE_CALL( hipEventDestroy(start) );
	CUDA_SAFE_CALL( hipEventDestroy(stop) );
	return kernel_time;
}

void runbench(int type, double* kernel_time, double* bandw){

	const long long shared_access = 2*(long long)(COMP_ITERATIONS)*THREADS*BLOCKS;

	dim3 dimBlock(THREADS, 1, 1);
    dim3 dimGrid(BLOCKS, 1, 1);
	hipEvent_t start, stop;

	initializeEvents(&start, &stop);
	benchmark<float><<< dimGrid, dimBlock >>>();

	hipDeviceSynchronize();

	double time = finalizeEvents(start, stop);
	double result;
	if (type==0)
		result = ((double)shared_access)*4/(double)time*1000./(double)(1000*1000*1000);
	else
		result = ((double)shared_access)*8/(double)time*1000./(double)(1000*1000*1000);

	*kernel_time = time;
	*bandw=result;

}


int main(int argc, char *argv[]){
	// CUpti_SubscriberHandle subscriber;
	hipDevice_t device = 0;
	int deviceCount;
	char deviceName[32];
	hipDeviceProp_t deviceProp;


	printf("Usage: %s [device_num] [metric_name]\n", argv[0]);
	int ntries;
	if (argc>1){
		ntries = atoi(argv[1]);
	}else{
		ntries = 1;
	}

	hipSetDevice(deviceNum);
	double time[ntries][2],value[ntries][4];

	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
		printf("There is no device supporting CUDA.\n");
	return -2;
	}

	printf("CUDA Device Number: %d\n", deviceNum);

	hipDeviceGet(&device, deviceNum);
	CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, device));
	hipDeviceGetName(deviceName, 32, device);

	// DRIVER_API_CALL(hipCtxCreate(&context, 0, device));
	int i;
	class type;

	int dodouble=0;
	for (i=0;i<ntries;i++){
		runbench(dodouble,&time[0][0],&value[0][0]);

		printf("Registered time: %f ms\n",time[0][0]);

	}


	CUDA_SAFE_CALL( hipDeviceReset());

	return 0;
}
