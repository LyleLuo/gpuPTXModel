#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
//#include <cutil.h>
#include <hip/hip_runtime.h>

float* h_A;
float* h_B;
float* h_C;
float* h_res;
float* d_A;
float* d_B;
float* d_C;
float* d_res;

__global__
//void compute(const float* A, const float* B, const float* C, float* D, int n) {
void compute(float* D, int n, int div) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    float I1 = tid * 2.0;

    int thread_id = threadIdx.x % 32;

    if (thread_id < div) {
        __asm volatile (
                " .reg .f32 %r1111;\n\t"
                " .reg .f32 %r1112;\n\t"
                " .reg .f32 %r1113;\n\t"
                " .reg .f32 %r1114;\n\t"
                " .reg .f32 %r1115;\n\t"
                " .reg .f32 %r1116;\n\t"
                " .reg .f32 %r1117;\n\t"
                " .reg .f32 %r1118;\n\t"
                " .reg .f32 %r1119;\n\t"
                " .reg .f32 %r1120;\n\t"
                " .reg .f32 %r1121;\n\t"
                " .reg .f32 %r1122;\n\t"
                " .reg .f32 %r1123;\n\t"
                " .reg .f32 %r1124;\n\t"
                " .reg .f32 %r1125;\n\t"
                " .reg .f32 %r1126;\n\t"
                " .reg .f32 %r1127;\n\t"
                " .reg .f32 %r1128;\n\t"
                "mov.f32 %r1112, 4.4;\n\t"
                "mov.f32 %r1113, %r1112;\n\t"
                "mov.f32 %r1114, 2.2;\n\t"
                "mov.f32 %r1115, 3.3;\n\t"
                "mov.f32 %r1116, 1.23;\n\t"
                "mov.f32 %r1117, 2.42;\n\t"
                "mov.f32 %r1118, 3.34;\n\t"
                "mov.f32 %r1119, 5.62;\n\t"
                "mov.f32 %r1120, 2.56;\n\t"
                "mov.f32 %r1121, 1.56;\n\t"
                "mov.f32 %r1122, 2.56;\n\t"
                "mov.f32 %r1123, 5.56;\n\t"
                "mov.f32 %r1124, 8.56;\n\t"
                "mov.f32 %r1125, 3.56;\n\t"
                "mov.f32 %r1126, 5.56;\n\t"
                "mov.f32 %r1127, 6.56;\n\t"
                "mov.f32 %r1128, 0.56;\n\t"
                );
        for (int k = 0; k < n; k++) {
            __asm volatile (
                    "fma.rn.f32 %r1113, %r1111, %r1113, %r1112;\n\t"
                    "fma.rn.f32 %r1114, %r1111, %r1114, %r1112;\n\t"
                    "fma.rn.f32 %r1115, %r1111, %r1115, %r1112;\n\t"
                    "fma.rn.f32 %r1116, %r1111, %r1116, %r1112;\n\t"
                    "fma.rn.f32 %r1117, %r1111, %r1117, %r1112;\n\t"
                    "fma.rn.f32 %r1118, %r1111, %r1118, %r1112;\n\t"
                    "fma.rn.f32 %r1119, %r1111, %r1119, %r1112;\n\t"
                    "fma.rn.f32 %r1120, %r1111, %r1120, %r1112;\n\t"
                    "fma.rn.f32 %r1121, %r1111, %r1121, %r1112;\n\t"
                    "fma.rn.f32 %r1122, %r1111, %r1122, %r1112;\n\t"
                    "fma.rn.f32 %r1123, %r1111, %r1123, %r1112;\n\t"
                    "fma.rn.f32 %r1124, %r1111, %r1124, %r1112;\n\t"
                    "fma.rn.f32 %r1125, %r1111, %r1125, %r1112;\n\t"
                    "fma.rn.f32 %r1126, %r1111, %r1126, %r1112;\n\t"
                    "fma.rn.f32 %r1127, %r1111, %r1127, %r1112;\n\t"
                    "fma.rn.f32 %r1128, %r1111, %r1128, %r1112;\n\t"
                    "fma.rn.f32 %r1113, %r1111, %r1113, %r1112;\n\t"
                    "fma.rn.f32 %r1114, %r1111, %r1114, %r1112;\n\t"
                    "fma.rn.f32 %r1115, %r1111, %r1115, %r1112;\n\t"
                    "fma.rn.f32 %r1116, %r1111, %r1116, %r1112;\n\t"
                    "fma.rn.f32 %r1117, %r1111, %r1117, %r1112;\n\t"
                    "fma.rn.f32 %r1118, %r1111, %r1118, %r1112;\n\t"
                    "fma.rn.f32 %r1119, %r1111, %r1119, %r1112;\n\t"
                    "fma.rn.f32 %r1120, %r1111, %r1120, %r1112;\n\t"
                    "fma.rn.f32 %r1121, %r1111, %r1121, %r1112;\n\t"
                    "fma.rn.f32 %r1122, %r1111, %r1122, %r1112;\n\t"
                    "fma.rn.f32 %r1123, %r1111, %r1123, %r1112;\n\t"
                    "fma.rn.f32 %r1124, %r1111, %r1124, %r1112;\n\t"
                    "fma.rn.f32 %r1125, %r1111, %r1125, %r1112;\n\t"
                    "fma.rn.f32 %r1126, %r1111, %r1126, %r1112;\n\t"
                    "fma.rn.f32 %r1127, %r1111, %r1127, %r1112;\n\t"
                    "fma.rn.f32 %r1128, %r1111, %r1128, %r1112;\n\t"
                    "fma.rn.f32 %r1113, %r1111, %r1113, %r1112;\n\t"
                    "fma.rn.f32 %r1114, %r1111, %r1114, %r1112;\n\t"
                    "fma.rn.f32 %r1115, %r1111, %r1115, %r1112;\n\t"
                    "fma.rn.f32 %r1116, %r1111, %r1116, %r1112;\n\t"
                    "fma.rn.f32 %r1117, %r1111, %r1117, %r1112;\n\t"
                    "fma.rn.f32 %r1118, %r1111, %r1118, %r1112;\n\t"
                    "fma.rn.f32 %r1119, %r1111, %r1119, %r1112;\n\t"
                    "fma.rn.f32 %r1120, %r1111, %r1120, %r1112;\n\t"
                    "fma.rn.f32 %r1121, %r1111, %r1121, %r1112;\n\t"
                    "fma.rn.f32 %r1122, %r1111, %r1122, %r1112;\n\t"
                    "fma.rn.f32 %r1123, %r1111, %r1123, %r1112;\n\t"
                    "fma.rn.f32 %r1124, %r1111, %r1124, %r1112;\n\t"
                    "fma.rn.f32 %r1125, %r1111, %r1125, %r1112;\n\t"
                    "fma.rn.f32 %r1126, %r1111, %r1126, %r1112;\n\t"
                    "fma.rn.f32 %r1127, %r1111, %r1127, %r1112;\n\t"
                    "fma.rn.f32 %r1128, %r1111, %r1128, %r1112;\n\t"
                    "fma.rn.f32 %r1113, %r1111, %r1113, %r1112;\n\t"
                    "fma.rn.f32 %r1114, %r1111, %r1114, %r1112;\n\t"
                    "fma.rn.f32 %r1115, %r1111, %r1115, %r1112;\n\t"
                    "fma.rn.f32 %r1116, %r1111, %r1116, %r1112;\n\t"
                    "fma.rn.f32 %r1117, %r1111, %r1117, %r1112;\n\t"
                    "fma.rn.f32 %r1118, %r1111, %r1118, %r1112;\n\t"
                    "fma.rn.f32 %r1119, %r1111, %r1119, %r1112;\n\t"
                    "fma.rn.f32 %r1120, %r1111, %r1120, %r1112;\n\t"
                    "fma.rn.f32 %r1121, %r1111, %r1121, %r1112;\n\t"
                    "fma.rn.f32 %r1122, %r1111, %r1122, %r1112;\n\t"
                    "fma.rn.f32 %r1123, %r1111, %r1123, %r1112;\n\t"
                    "fma.rn.f32 %r1124, %r1111, %r1124, %r1112;\n\t"
                    "fma.rn.f32 %r1125, %r1111, %r1125, %r1112;\n\t"
                    "fma.rn.f32 %r1126, %r1111, %r1126, %r1112;\n\t"
                    "fma.rn.f32 %r1127, %r1111, %r1127, %r1112;\n\t"
                    "fma.rn.f32 %r1128, %r1111, %r1128, %r1112;\n\t"
                    "fma.rn.f32 %r1113, %r1111, %r1113, %r1112;\n\t"
                    "fma.rn.f32 %r1114, %r1111, %r1114, %r1112;\n\t"
                    "fma.rn.f32 %r1115, %r1111, %r1115, %r1112;\n\t"
                    "fma.rn.f32 %r1116, %r1111, %r1116, %r1112;\n\t"
                    "fma.rn.f32 %r1117, %r1111, %r1117, %r1112;\n\t"
                    "fma.rn.f32 %r1118, %r1111, %r1118, %r1112;\n\t"
                    "fma.rn.f32 %r1119, %r1111, %r1119, %r1112;\n\t"
                    "fma.rn.f32 %r1120, %r1111, %r1120, %r1112;\n\t"
                    "fma.rn.f32 %r1121, %r1111, %r1121, %r1112;\n\t"
                    "fma.rn.f32 %r1122, %r1111, %r1122, %r1112;\n\t"
                    "fma.rn.f32 %r1123, %r1111, %r1123, %r1112;\n\t"
                    "fma.rn.f32 %r1124, %r1111, %r1124, %r1112;\n\t"
                    "fma.rn.f32 %r1125, %r1111, %r1125, %r1112;\n\t"
                    "fma.rn.f32 %r1126, %r1111, %r1126, %r1112;\n\t"
                    "fma.rn.f32 %r1127, %r1111, %r1127, %r1112;\n\t"
                    "fma.rn.f32 %r1128, %r1111, %r1128, %r1112;\n\t"
                    "fma.rn.f32 %r1113, %r1111, %r1113, %r1112;\n\t"
                    "fma.rn.f32 %r1114, %r1111, %r1114, %r1112;\n\t"
                    "fma.rn.f32 %r1115, %r1111, %r1115, %r1112;\n\t"
                    "fma.rn.f32 %r1116, %r1111, %r1116, %r1112;\n\t"
                    "fma.rn.f32 %r1117, %r1111, %r1117, %r1112;\n\t"
                    "fma.rn.f32 %r1118, %r1111, %r1118, %r1112;\n\t"
                    "fma.rn.f32 %r1119, %r1111, %r1119, %r1112;\n\t"
                    "fma.rn.f32 %r1120, %r1111, %r1120, %r1112;\n\t"
                    "fma.rn.f32 %r1121, %r1111, %r1121, %r1112;\n\t"
                    "fma.rn.f32 %r1122, %r1111, %r1122, %r1112;\n\t"
                    "fma.rn.f32 %r1123, %r1111, %r1123, %r1112;\n\t"
                    "fma.rn.f32 %r1124, %r1111, %r1124, %r1112;\n\t"
                    "fma.rn.f32 %r1125, %r1111, %r1125, %r1112;\n\t"
                    "fma.rn.f32 %r1126, %r1111, %r1126, %r1112;\n\t"
                    "fma.rn.f32 %r1127, %r1111, %r1127, %r1112;\n\t"
                    "fma.rn.f32 %r1128, %r1111, %r1128, %r1112;\n\t"
                    "fma.rn.f32 %r1113, %r1111, %r1113, %r1112;\n\t"
                    "fma.rn.f32 %r1114, %r1111, %r1114, %r1112;\n\t"
                    "fma.rn.f32 %r1115, %r1111, %r1115, %r1112;\n\t"
                    "fma.rn.f32 %r1116, %r1111, %r1116, %r1112;\n\t"
                    "fma.rn.f32 %r1117, %r1111, %r1117, %r1112;\n\t"
                    "fma.rn.f32 %r1118, %r1111, %r1118, %r1112;\n\t"
                    "fma.rn.f32 %r1119, %r1111, %r1119, %r1112;\n\t"
                    "fma.rn.f32 %r1120, %r1111, %r1120, %r1112;\n\t"
                    "fma.rn.f32 %r1121, %r1111, %r1121, %r1112;\n\t"
                    "fma.rn.f32 %r1122, %r1111, %r1122, %r1112;\n\t"
                    "fma.rn.f32 %r1123, %r1111, %r1123, %r1112;\n\t"
                    "fma.rn.f32 %r1124, %r1111, %r1124, %r1112;\n\t"
                    "fma.rn.f32 %r1125, %r1111, %r1125, %r1112;\n\t"
                    "fma.rn.f32 %r1126, %r1111, %r1126, %r1112;\n\t"
                    "fma.rn.f32 %r1127, %r1111, %r1127, %r1112;\n\t"
                    "fma.rn.f32 %r1128, %r1111, %r1128, %r1112;\n\t"
                    "fma.rn.f32 %r1113, %r1111, %r1113, %r1112;\n\t"
                    "fma.rn.f32 %r1114, %r1111, %r1114, %r1112;\n\t"
                    "fma.rn.f32 %r1115, %r1111, %r1115, %r1112;\n\t"
                    "fma.rn.f32 %r1116, %r1111, %r1116, %r1112;\n\t"
                    "fma.rn.f32 %r1117, %r1111, %r1117, %r1112;\n\t"
                    "fma.rn.f32 %r1118, %r1111, %r1118, %r1112;\n\t"
                    "fma.rn.f32 %r1119, %r1111, %r1119, %r1112;\n\t"
                    "fma.rn.f32 %r1120, %r1111, %r1120, %r1112;\n\t"
                    "fma.rn.f32 %r1121, %r1111, %r1121, %r1112;\n\t"
                    "fma.rn.f32 %r1122, %r1111, %r1122, %r1112;\n\t"
                    "fma.rn.f32 %r1123, %r1111, %r1123, %r1112;\n\t"
                    "fma.rn.f32 %r1124, %r1111, %r1124, %r1112;\n\t"
                    "fma.rn.f32 %r1125, %r1111, %r1125, %r1112;\n\t"
                    "fma.rn.f32 %r1126, %r1111, %r1126, %r1112;\n\t"
                    "fma.rn.f32 %r1127, %r1111, %r1127, %r1112;\n\t"
                    "fma.rn.f32 %r1128, %r1111, %r1128, %r1112;\n\t"
                    "fma.rn.f32 %r1113, %r1111, %r1113, %r1112;\n\t"
                    "fma.rn.f32 %r1114, %r1111, %r1114, %r1112;\n\t"
                    "fma.rn.f32 %r1115, %r1111, %r1115, %r1112;\n\t"
                    "fma.rn.f32 %r1116, %r1111, %r1116, %r1112;\n\t"
                    "fma.rn.f32 %r1117, %r1111, %r1117, %r1112;\n\t"
                    "fma.rn.f32 %r1118, %r1111, %r1118, %r1112;\n\t"
                    "fma.rn.f32 %r1119, %r1111, %r1119, %r1112;\n\t"
                    "fma.rn.f32 %r1120, %r1111, %r1120, %r1112;\n\t"
                    "fma.rn.f32 %r1121, %r1111, %r1121, %r1112;\n\t"
                    "fma.rn.f32 %r1122, %r1111, %r1122, %r1112;\n\t"
                    "fma.rn.f32 %r1123, %r1111, %r1123, %r1112;\n\t"
                    "fma.rn.f32 %r1124, %r1111, %r1124, %r1112;\n\t"
                    "fma.rn.f32 %r1125, %r1111, %r1125, %r1112;\n\t"
                    "fma.rn.f32 %r1126, %r1111, %r1126, %r1112;\n\t"
                    "fma.rn.f32 %r1127, %r1111, %r1127, %r1112;\n\t"
                    "fma.rn.f32 %r1128, %r1111, %r1128, %r1112;\n\t"
                    "fma.rn.f32 %r1113, %r1111, %r1113, %r1112;\n\t"
                    "fma.rn.f32 %r1114, %r1111, %r1114, %r1112;\n\t"
                    "fma.rn.f32 %r1115, %r1111, %r1115, %r1112;\n\t"
                    "fma.rn.f32 %r1116, %r1111, %r1116, %r1112;\n\t"
                    "fma.rn.f32 %r1117, %r1111, %r1117, %r1112;\n\t"
                    "fma.rn.f32 %r1118, %r1111, %r1118, %r1112;\n\t"
                    "fma.rn.f32 %r1119, %r1111, %r1119, %r1112;\n\t"
                    "fma.rn.f32 %r1120, %r1111, %r1120, %r1112;\n\t"
                    "fma.rn.f32 %r1121, %r1111, %r1121, %r1112;\n\t"
                    "fma.rn.f32 %r1122, %r1111, %r1122, %r1112;\n\t"
                    "fma.rn.f32 %r1123, %r1111, %r1123, %r1112;\n\t"
                    "fma.rn.f32 %r1124, %r1111, %r1124, %r1112;\n\t"
                    "fma.rn.f32 %r1125, %r1111, %r1125, %r1112;\n\t"
                    "fma.rn.f32 %r1126, %r1111, %r1126, %r1112;\n\t"
                    "fma.rn.f32 %r1127, %r1111, %r1127, %r1112;\n\t"
                    "fma.rn.f32 %r1128, %r1111, %r1128, %r1112;\n\t"
                    "fma.rn.f32 %r1113, %r1111, %r1113, %r1112;\n\t"
                    "fma.rn.f32 %r1114, %r1111, %r1114, %r1112;\n\t"
                    "fma.rn.f32 %r1115, %r1111, %r1115, %r1112;\n\t"
                    "fma.rn.f32 %r1116, %r1111, %r1116, %r1112;\n\t"
                    "fma.rn.f32 %r1117, %r1111, %r1117, %r1112;\n\t"
                    "fma.rn.f32 %r1118, %r1111, %r1118, %r1112;\n\t"
                    "fma.rn.f32 %r1119, %r1111, %r1119, %r1112;\n\t"
                    "fma.rn.f32 %r1120, %r1111, %r1120, %r1112;\n\t"
                    "fma.rn.f32 %r1121, %r1111, %r1121, %r1112;\n\t"
                    "fma.rn.f32 %r1122, %r1111, %r1122, %r1112;\n\t"
                    "fma.rn.f32 %r1123, %r1111, %r1123, %r1112;\n\t"
                    "fma.rn.f32 %r1124, %r1111, %r1124, %r1112;\n\t"
                    "fma.rn.f32 %r1125, %r1111, %r1125, %r1112;\n\t"
                    "fma.rn.f32 %r1126, %r1111, %r1126, %r1112;\n\t"
                    "fma.rn.f32 %r1127, %r1111, %r1127, %r1112;\n\t"
                    "fma.rn.f32 %r1128, %r1111, %r1128, %r1112;\n\t"
                    "fma.rn.f32 %r1113, %r1111, %r1113, %r1112;\n\t"
                    "fma.rn.f32 %r1114, %r1111, %r1114, %r1112;\n\t"
                    "fma.rn.f32 %r1115, %r1111, %r1115, %r1112;\n\t"
                    "fma.rn.f32 %r1116, %r1111, %r1116, %r1112;\n\t"
                    "fma.rn.f32 %r1117, %r1111, %r1117, %r1112;\n\t"
                    "fma.rn.f32 %r1118, %r1111, %r1118, %r1112;\n\t"
                    "fma.rn.f32 %r1119, %r1111, %r1119, %r1112;\n\t"
                    "fma.rn.f32 %r1120, %r1111, %r1120, %r1112;\n\t"
                    "fma.rn.f32 %r1121, %r1111, %r1121, %r1112;\n\t"
                    "fma.rn.f32 %r1122, %r1111, %r1122, %r1112;\n\t"
                    "fma.rn.f32 %r1123, %r1111, %r1123, %r1112;\n\t"
                    "fma.rn.f32 %r1124, %r1111, %r1124, %r1112;\n\t"
                    "fma.rn.f32 %r1125, %r1111, %r1125, %r1112;\n\t"
                    "fma.rn.f32 %r1126, %r1111, %r1126, %r1112;\n\t"
                    "fma.rn.f32 %r1127, %r1111, %r1127, %r1112;\n\t"
                    "fma.rn.f32 %r1128, %r1111, %r1128, %r1112;\n\t"
                    "fma.rn.f32 %r1113, %r1111, %r1113, %r1112;\n\t"
                    "fma.rn.f32 %r1114, %r1111, %r1114, %r1112;\n\t"
                    "fma.rn.f32 %r1115, %r1111, %r1115, %r1112;\n\t"
                    "fma.rn.f32 %r1116, %r1111, %r1116, %r1112;\n\t"
                    "fma.rn.f32 %r1117, %r1111, %r1117, %r1112;\n\t"
                    "fma.rn.f32 %r1118, %r1111, %r1118, %r1112;\n\t"
                    "fma.rn.f32 %r1119, %r1111, %r1119, %r1112;\n\t"
                    "fma.rn.f32 %r1120, %r1111, %r1120, %r1112;\n\t"
                    "fma.rn.f32 %r1121, %r1111, %r1121, %r1112;\n\t"
                    "fma.rn.f32 %r1122, %r1111, %r1122, %r1112;\n\t"
                    "fma.rn.f32 %r1123, %r1111, %r1123, %r1112;\n\t"
                    "fma.rn.f32 %r1124, %r1111, %r1124, %r1112;\n\t"
                    "fma.rn.f32 %r1125, %r1111, %r1125, %r1112;\n\t"
                    "fma.rn.f32 %r1126, %r1111, %r1126, %r1112;\n\t"
                    "fma.rn.f32 %r1127, %r1111, %r1127, %r1112;\n\t"
                    "fma.rn.f32 %r1128, %r1111, %r1128, %r1112;\n\t"
                    "fma.rn.f32 %r1113, %r1111, %r1113, %r1112;\n\t"
                    "fma.rn.f32 %r1114, %r1111, %r1114, %r1112;\n\t"
                    "fma.rn.f32 %r1115, %r1111, %r1115, %r1112;\n\t"
                    "fma.rn.f32 %r1116, %r1111, %r1116, %r1112;\n\t"
                    "fma.rn.f32 %r1117, %r1111, %r1117, %r1112;\n\t"
                    "fma.rn.f32 %r1118, %r1111, %r1118, %r1112;\n\t"
                    "fma.rn.f32 %r1119, %r1111, %r1119, %r1112;\n\t"
                    "fma.rn.f32 %r1120, %r1111, %r1120, %r1112;\n\t"
                    "fma.rn.f32 %r1121, %r1111, %r1121, %r1112;\n\t"
                    "fma.rn.f32 %r1122, %r1111, %r1122, %r1112;\n\t"
                    "fma.rn.f32 %r1123, %r1111, %r1123, %r1112;\n\t"
                    "fma.rn.f32 %r1124, %r1111, %r1124, %r1112;\n\t"
                    "fma.rn.f32 %r1125, %r1111, %r1125, %r1112;\n\t"
                    "fma.rn.f32 %r1126, %r1111, %r1126, %r1112;\n\t"
                    "fma.rn.f32 %r1127, %r1111, %r1127, %r1112;\n\t"
                    "fma.rn.f32 %r1128, %r1111, %r1128, %r1112;\n\t"
                    "fma.rn.f32 %r1113, %r1111, %r1113, %r1112;\n\t"
                    "fma.rn.f32 %r1114, %r1111, %r1114, %r1112;\n\t"
                    "fma.rn.f32 %r1115, %r1111, %r1115, %r1112;\n\t"
                    "fma.rn.f32 %r1116, %r1111, %r1116, %r1112;\n\t"
                    "fma.rn.f32 %r1117, %r1111, %r1117, %r1112;\n\t"
                    "fma.rn.f32 %r1118, %r1111, %r1118, %r1112;\n\t"
                    "fma.rn.f32 %r1119, %r1111, %r1119, %r1112;\n\t"
                    "fma.rn.f32 %r1120, %r1111, %r1120, %r1112;\n\t"
                    "fma.rn.f32 %r1121, %r1111, %r1121, %r1112;\n\t"
                    "fma.rn.f32 %r1122, %r1111, %r1122, %r1112;\n\t"
                    "fma.rn.f32 %r1123, %r1111, %r1123, %r1112;\n\t"
                    "fma.rn.f32 %r1124, %r1111, %r1124, %r1112;\n\t"
                    "fma.rn.f32 %r1125, %r1111, %r1125, %r1112;\n\t"
                    "fma.rn.f32 %r1126, %r1111, %r1126, %r1112;\n\t"
                    "fma.rn.f32 %r1127, %r1111, %r1127, %r1112;\n\t"
                    "fma.rn.f32 %r1128, %r1111, %r1128, %r1112;\n\t"
                    "fma.rn.f32 %r1113, %r1111, %r1113, %r1112;\n\t"
                    "fma.rn.f32 %r1114, %r1111, %r1114, %r1112;\n\t"
                    "fma.rn.f32 %r1115, %r1111, %r1115, %r1112;\n\t"
                    "fma.rn.f32 %r1116, %r1111, %r1116, %r1112;\n\t"
                    "fma.rn.f32 %r1117, %r1111, %r1117, %r1112;\n\t"
                    "fma.rn.f32 %r1118, %r1111, %r1118, %r1112;\n\t"
                    "fma.rn.f32 %r1119, %r1111, %r1119, %r1112;\n\t"
                    "fma.rn.f32 %r1120, %r1111, %r1120, %r1112;\n\t"
                    "fma.rn.f32 %r1121, %r1111, %r1121, %r1112;\n\t"
                    "fma.rn.f32 %r1122, %r1111, %r1122, %r1112;\n\t"
                    "fma.rn.f32 %r1123, %r1111, %r1123, %r1112;\n\t"
                    "fma.rn.f32 %r1124, %r1111, %r1124, %r1112;\n\t"
                    "fma.rn.f32 %r1125, %r1111, %r1125, %r1112;\n\t"
                    "fma.rn.f32 %r1126, %r1111, %r1126, %r1112;\n\t"
                    "fma.rn.f32 %r1127, %r1111, %r1127, %r1112;\n\t"
                    "fma.rn.f32 %r1128, %r1111, %r1128, %r1112;\n\t"
                    );
        }
    }
    __syncthreads();

    //    if ((blockDim.x * blockIdx.x + threadIdx.x) == 0)
    *D = I1;

    __syncthreads();
}

void usage() {
    std::cout << "Usage ./binary <num_blocks> <num_threads_per_block> <iterations>" "threads active per warp" << std::endl;
}

int main(int argc, char **argv)
{
    if (argc != 5) {
        usage();
        exit(1);
    }

    int num_blocks = atoi(argv[1]);
    int num_threads_per_block = atoi(argv[2]);
    int iterations = atoi(argv[3]);
    int divergence = atoi(argv[4]);

//    h_A = new float(2.0);
//    h_B = new float(3.0);
//    h_C = new float(4.0);

//    cudaMalloc((void**)&d_A, sizeof(float));
//    cudaMalloc((void**)&d_B, sizeof(float));
//    cudaMalloc((void**)&d_C, sizeof(float));
    hipMalloc((void**)&d_res, sizeof(float));

//    cudaMemcpy(d_A, h_A, sizeof(float), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_B, h_B, sizeof(float), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_C, h_C, sizeof(float), cudaMemcpyHostToDevice);

    hipEvent_t start, stop;
    float time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    hipProfilerStart();

//    compute<<<num_blocks, num_threads_per_block>>>(d_A, d_B, d_C, d_res, iterations);
    compute<<<num_blocks, num_threads_per_block>>>(d_res, iterations, divergence);

    hipProfilerStop();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);

    std::cout << "GPU Elapsed Time = " << time << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceSynchronize();

    hipMemcpy(h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);

    return 0;
}
